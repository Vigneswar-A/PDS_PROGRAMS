#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 25

__global__ void dot_product(int *A, int *B, int *C)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;
    C[idx] = A[idx] * B[idx];  
}

__global__ void reduce(int *C)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;
    // Parallel reduction technique
    int stride = 1;
    while (stride < N && idx%stride == 0){
        stride *= 2;
        int parent = idx/stride * stride;
        if (parent != idx){
            C[parent] += C[idx];
        }
        __syncthreads();
    }
}

int main()
{
    // Create host and device variables
    int A[N], B[N], C[N];
    int *devA, *devB, *devC;

    // Initialize array with sample input values
    for (int i = 0; i < N; i++)
    {
        A[i] = 1;
        B[i] = 2;
    }

    // Allocate device memory
    hipMalloc(&devA, sizeof(A));
    hipMalloc(&devB, sizeof(B));
    hipMalloc(&devC, sizeof(C));

    // Copy arrays to device
    hipMemcpy(devA, A, sizeof(A), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, sizeof(B), hipMemcpyHostToDevice);

    // Initialize events to compute execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Compute the dot product
    hipEventRecord(start);
    dot_product<<<1, N>>>(devA, devB, devC);
    reduce<<<1, N>>>(devC);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Copy result to host memory
    hipMemcpy(C, devC, sizeof(C), hipMemcpyDeviceToHost);

    // Compute execution time
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    printf("Dot product : %d\n", C[0]);
    printf("Execution time (CUDA): %fms\n", ms);

    // Free the device memory
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);

    return 0;
}

/*
Dot product : 50
Execution time (CUDA): 0.037312ms
*/
